#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include "common/catch.hpp"
#define CATCH_CONFIG_CPP11_TO_STRING
#define CATCH_CONFIG_COLOUR_ANSI
#include "seqdata_permute.cuh"


template <typename T>
class test_SeqData{

public:
  test_SeqData(std::array<int,4> dimA, std::array<int,4> permuteA, unsigned mode){
    this->dimA = dimA;
    this->permuteA = permuteA;
    this->mode = mode;

    data_len = dimA[0]*dimA[1]*dimA[2]*dimA[3];
    input = std::vector<float>(data_len);
    output = std::vector<float>(data_len, 0.0f);

    
    // allocate memory for h_input and h_output
    h_input = malloc(data_len*sizeof(T));
    h_output = malloc(data_len*sizeof(T));
  }
  ~test_SeqData(){
    free(h_input);
    free(h_output);
  }

  void init_data() {

    std::mt19937 rng = std::mt19937(2023);
    std::uniform_real_distribution<float> uf_distribution = std::uniform_real_distribution<float>(-10, 10);

    std::generate(std::begin(input), std::end(input), [&]{return uf_distribution(rng);} ); 

    // copy the input to h_input data by casting float to type T
    std::transform(std::begin(input), std::end(input), reinterpret_cast<T*>(h_input), [](float x){return static_cast<T>(x);});
  }

  
  void run_gpu_permute() {
    // deivce ptr
    T *d_input, *d_output;
    
    CHECK_CUDA_ERR(hipMalloc((void**)&d_input, data_len*sizeof(T)));
    CHECK_CUDA_ERR(hipMalloc((void**)&d_output, data_len*sizeof(T)));

    // copy h2d
    CHECK_CUDA_ERR(hipMemcpy(d_input,h_input,data_len*sizeof(T),hipMemcpyHostToDevice));

    hipStream_t stream;
    CHECK_CUDA_ERR(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    LaunchTransformSeqDataAxesKernel<T>(stream, mode, dimA.data(), permuteA.data(), d_input, d_output);
    CHECK_CUDA_ERR(hipStreamSynchronize(stream));


    // copy d2h
    CHECK_CUDA_ERR(hipMemcpy(h_output,d_output,data_len*sizeof(T),hipMemcpyDeviceToHost));

    CHECK_CUDA_ERR(hipFree(d_input));
    CHECK_CUDA_ERR(hipFree(d_output));
  }

  void run_cpu_permute() {

    
    std::function<int(int)> dim_factorial = [this,&dim_factorial](int n) -> int {
      assert(n<=3);
      if (n == 3) return 1;
      return dimA[n+1] * dim_factorial(n + 1);
    };
    
    int stride_0 = dim_factorial(0);
    int stride_1 = dim_factorial(1);
    int stride_2 = dim_factorial(2);

    int perm_stride_0, perm_stride_1, perm_stride_2;
    if(mode == 0){
      std::function<int(int)> perm_dim_factorial = [this,&perm_dim_factorial](int n) -> int {
        assert(n<=3);
        if (n == 3) return 1;
        int np = std::find(permuteA.begin(), permuteA.end(), n+1) - permuteA.begin();
        return dimA[np] * perm_dim_factorial(permuteA[np]);
      };
      
      perm_stride_0 = perm_dim_factorial(permuteA[0]);
      perm_stride_1 = perm_dim_factorial(permuteA[1]);
      perm_stride_2 = perm_dim_factorial(permuteA[2]);
    }
    else if(mode == 1){
      std::function<int(int)> perm_dim_factorial = [this,&perm_dim_factorial](int n) -> int {
        assert(n<=3);
        if (n == 3) return 1;
        int np = std::find(permuteA.begin(), permuteA.end(), n) - permuteA.begin();
        return dimA[permuteA[np+1]] * perm_dim_factorial(permuteA[np+1]);
      };
      
      perm_stride_0 = perm_dim_factorial(0);
      perm_stride_1 = perm_dim_factorial(1);
      perm_stride_2 = perm_dim_factorial(2);

      std::cout << "perm_stride_0: " << perm_stride_0 << std::endl;
      std::cout << "perm_stride_1: " << perm_stride_1 << std::endl;
      std::cout << "perm_stride_2: " << perm_stride_2 << std::endl;
    }
    else{
      assert(false);
    }

    // cpu implementation
    for (int n = 0; n < dimA[0]; n++) {
      for (int c = 0; c < dimA[1]; c++) {
        for (int h = 0; h < dimA[2]; h++) {
          for (int w = 0; w < dimA[3]; w++) {
            int in_idx = n * stride_0 + c * stride_1 + h * stride_2 + w;
            int out_idx = n * perm_stride_0 + c * perm_stride_1 + h * perm_stride_2 + w;
            output[out_idx] = input[in_idx];
          }
        }
      }
    }
  }


  void print_vec(const std::vector<float> outv, std::string outn, int start = 0) {
      std::cout << outn << ": ";
      std::copy(outv.begin() + start, outv.begin() + ((start + 64)>outv.size()?outv.size():(start + 64)), std::ostream_iterator<float>(std::cout, ", "));
      std::cout << std::endl;
    }

  void verify() {
      // print_vec(input, "input");
      // print_vec(output, "output");
      SECTION(std::string(ANSI_COLOR_RED) + std::to_string(data_len) + ANSI_COLOR_RESET) {
        bool is_near2 = true;
        size_t count = 0;
        for (int i = 0; i < output.size(); i++) {
          bool is_this_near2 = NEAR2(static_cast<float>(reinterpret_cast<T*>(h_output)[i]), output[i], 1e-2);
          if(!is_this_near2 && count<64){
            count++;
            fmt::print(ANSI_COLOR_RED "ERROR @ {}[{}] {} vs {}\n" ANSI_COLOR_RESET, std::to_string(data_len), i, static_cast<float>(reinterpret_cast<T*>(h_output)[i]), output[i]);
          }
          is_near2 &= is_this_near2;
        }
        CHECK(is_near2);
      }
    }


private:
  std::array<int,4> dimA, permuteA;
  size_t data_len;
  std::vector<float> input, output;
  void *h_input, *h_output;
  unsigned mode;

  std::function<bool(float,float,float)> NEAR2 = [](float a, float b, float prec) -> bool { return ((a != a && b != b) 
      || (a == std::numeric_limits<typename std::remove_reference<decltype(a)>::type>::infinity() 
        && b == std::numeric_limits<typename std::remove_reference<  decltype(b)>::type>::infinity()) 
      || (-a == std::numeric_limits<typename std::remove_reference< decltype(a)>::type>::infinity() 
        && -b == std::numeric_limits<typename std::remove_reference<  decltype(b)>::type>::infinity()) 
      || (abs(a - b) / abs(a) < prec) || (abs(a - b) / abs(b) < prec) || (abs(a - b) < prec)); };

};

template <typename T>
int eval_seqdata(const std::array<int,4>& dim_a, const std::array<int,4>& permute_a, const unsigned mode){
  test_SeqData<T> test_seqdata(dim_a, permute_a, mode);
  test_seqdata.init_data();
  test_seqdata.run_gpu_permute();
  test_seqdata.run_cpu_permute();
  test_seqdata.verify();
}

TEST_CASE("SeqData", "[SeqData]") {
  SECTION("1") {
    eval_seqdata<float>({16,32,64,512},{2,1,0,3},0);
  }
  SECTION("2") {
    eval_seqdata<__half>({16,32,64,512},{2,1,0,3},0);
  }

  SECTION("3") {
    eval_seqdata<float>({16,32,64,512},{2,0,1,3},0);
  }
  SECTION("4") {
    eval_seqdata<__half>({16,32,64,512},{2,0,1,3},0);
  }
  
  SECTION("5") {
    eval_seqdata<float>({16,32,64,512},{0,2,1,3},0);
  }
  SECTION("6") {
    eval_seqdata<__half>({16,32,64,512},{0,2,1,3},0);
  }

  SECTION("7") {
    eval_seqdata<float>({16,32,64,512},{1,2,0,3},0);
  }
  SECTION("8") {
    eval_seqdata<__half>({16,32,64,512},{1,2,0,3},0);
  }

  SECTION("9") {
    eval_seqdata<float>({16,32,64,512},{1,0,2,3},0);
  }
  SECTION("10") {
    eval_seqdata<__half>({16,32,64,512},{1,0,2,3},0);
  }


  SECTION("11") {
    eval_seqdata<float>({16,32,64,512},{2,1,0,3},1);
  }
  SECTION("12") {
    eval_seqdata<__half>({16,32,64,512},{2,1,0,3},1);
  }

  SECTION("13") {
    eval_seqdata<float>({16,32,64,512},{2,0,1,3},1);
  }
  SECTION("14") {
    eval_seqdata<__half>({16,32,64,512},{2,0,1,3},1);
  }
  
  SECTION("15") {
    eval_seqdata<float>({16,32,64,512},{0,2,1,3},1);
  }
  SECTION("16") {
    eval_seqdata<__half>({16,32,64,512},{0,2,1,3},1);
  }

  SECTION("17") {
    eval_seqdata<float>({16,32,64,512},{1,2,0,3},1);
  }
  SECTION("18") {
    eval_seqdata<__half>({16,32,64,512},{1,2,0,3},1);
  }

  SECTION("19") {
    eval_seqdata<float>({16,32,64,512},{1,0,2,3},1);
  }
  SECTION("20") {
    eval_seqdata<__half>({16,32,64,512},{1,0,2,3},1);
  }
}